#include "hip/hip_runtime.h"
﻿/*
Fast Lee Hologram computation using CUDA
Programmed by Shay Ohayon
DiCarlo Lab @ MIT

Revision History
Version 0.1 10/22/2014  
*/

#include <stdio.h>
#include "mex.h"
#include <Windows.h>
#include <math.h>

#define MIN(a,b) (a)<(b)?(a):(b)
#define M_PI 3.14159265358979323846

const int DMDwidth = 1024;
const int DMDheight = 768;
const int effectiveDMDwidth = DMDheight;



__global__ void computeCuda(double *inputPhases, bool *binaryPatterns, double *carrierWave, int patternSizeX, int patternSizeY, int numReferencePixels, int leeBlockSize) {
	int z = blockDim.x * blockIdx.x + threadIdx.x;

	long long output_offset = DMDwidth*DMDheight*z;
	long long input_offset = patternSizeX*patternSizeY*z;
	double phaseAngle = 0;


	for (int x = 0; x < DMDwidth; x++)
	{
		int sampleX = (x - numReferencePixels) / leeBlockSize;
		for (int y = 0; y < DMDheight; y++)
		{

			phaseAngle = 0.0;  //good

			if (y >= numReferencePixels && y < DMDheight - numReferencePixels && x >= numReferencePixels && x < effectiveDMDwidth - numReferencePixels)
			{
				int sampleY = (y - numReferencePixels) / leeBlockSize;
				assert(sampleX >= 0 && sampleY >= 0 && sampleX < patternSizeX && sampleY < patternSizeY);
				phaseAngle = inputPhases[input_offset + sampleX*patternSizeY + sampleY];
			}
			binaryPatterns[output_offset + x*DMDheight + y] = (0.5 * (1 + cos(carrierWave[x*DMDheight + y] - phaseAngle))) > 0.5;
		}
	}
}

void compute(int z, double *inputPhases, bool *binaryPatterns, double *carrierWave, int patternSizeX, int patternSizeY, int numReferencePixels, int leeBlockSize)
{

	long long output_offset = DMDwidth*DMDheight*z;
	long long input_offset = patternSizeX*patternSizeY*z;
	double phaseAngle = 0;


	for (int x = 0; x < DMDwidth; x++)
	{
		int sampleX = (x - numReferencePixels) / leeBlockSize;
		for (int y = 0; y < DMDheight; y++)
		{

			phaseAngle = 0.0;

			if (y >= numReferencePixels && y < DMDheight - numReferencePixels && x >= numReferencePixels && x < effectiveDMDwidth - numReferencePixels)
			{
				int sampleY = (y - numReferencePixels) / leeBlockSize;
				assert(sampleX >= 0 && sampleY >= 0 && sampleX < patternSizeX && sampleY < patternSizeY);
				phaseAngle = inputPhases[input_offset + sampleX*patternSizeY + sampleY];
			}
			binaryPatterns[output_offset + x*DMDheight + y] = (0.5 * (1 + cos(carrierWave[x*DMDheight + y] - phaseAngle))) > 0.5;
		}
	}
}


void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, const mxArray *prhs[]) {

	if (nrhs < 3 || nlhs != 1)
	{
		mexPrintf("Use: OutputBinaryPatterns = FastLeeHologram(inputPhases (NxNxM), numReferencePixels, leeBlockSize, selectedCarrier);");
		return;
	}

	double *inputPhases = (double*) mxGetData(prhs[0]);
	int numReferencePixels = *(double*)mxGetData(prhs[1]);
	int leeBlockSize = *(double*)mxGetData(prhs[2]);
	double selectedCarrier = *(double*) mxGetData(prhs[3]);

	const int numDim = mxGetNumberOfDimensions(prhs[0]);
	const int *dataSize = mxGetDimensions(prhs[0]);
	int numPatterns = 1;
	int patternSizeX = dataSize[0];
	int patternSizeY = dataSize[1];
	if (numDim > 2) 
	{
		numPatterns = dataSize[2];
	}

	// allocate memory for output
	const int outputDimSize[3] = { DMDheight, DMDwidth, numPatterns };
	plhs[0] = mxCreateLogicalArray(3, outputDimSize);
	bool* binaryPatterns = (bool*)mxGetData(plhs[0]);


	// allocate memory for the reference wave
	double *carrierWave = new double[DMDheight*DMDwidth];
	for (int x = 0; x < DMDwidth; x++)
	{
		for (int y = 0; y < DMDheight; y++)
		{
			carrierWave[x*DMDheight +y] = 2.0 * M_PI*(x - y)*selectedCarrier;
		}
	}
	

	double* d_inputPhases;
	long inputSize = sizeof(double) * patternSizeX * patternSizeY * numPatterns;
	hipMalloc(&d_inputPhases, inputSize);
	hipMemcpy(d_inputPhases, inputPhases, inputSize, hipMemcpyHostToDevice);


	int maxThreadsPerBlock = 256;
	int numBlocks = numPatterns / maxThreadsPerBlock;
	computeCuda << <numBlocks, maxThreadsPerBlock >> >(inputPhases, binaryPatterns, carrierWave, patternSizeX, patternSizeY, numReferencePixels, leeBlockSize);

	/*
	for (int z = 0; z < numPatterns; z++)
	{
		compute(z, inputPhases, binaryPatterns, carrierWave, patternSizeX, patternSizeY, numReferencePixels, leeBlockSize);
	}
	*/

	delete carrierWave;
}

